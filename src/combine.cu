#include <hip/hip_runtime.h>
#include <assert.h>
#include <iostream>
#include <sstream>
#include <fstream>

#define BLOCK_DIM 1024
#define MAX_DIMS 10
#define TILE 32

#define ADD_FUNC       1
#define MUL_FUNC       2
#define ID_FUNC        3
#define NEG_FUNC       4
#define LT_FUNC        5
#define EQ_FUNC        6
#define SIGMOID_FUNC   7
#define RELU_FUNC      8
#define RELU_BACK_FUNC 9
#define LOG_FUNC       10
#define LOG_BACK_FUNC  11
#define EXP_FUNC       12
#define INV_FUNC       13
#define INV_BACK_FUNC  14
#define IS_CLOSE_FUNC  15
#define MAX_FUNC       16
#define POW            17
#define TANH           18

__device__ float fn(int fn_id, float x, float y=0) {
    switch(fn_id) {
      case ADD_FUNC: {
        return x + y;
      }
      case MUL_FUNC: {
        return x * y;
      }
      case ID_FUNC: {
      	return x;
      }
      case NEG_FUNC: {
        return -x;
      }
      case LT_FUNC: {
        if (x < y) {
          return 1.0;
        }
        else {
          return 0.0;
        }
      }
      case EQ_FUNC: {
        if (x == y) {
          return 1.0;
        }
        else {
          return 0.0;
        }
      }
      case SIGMOID_FUNC: {
        if (x >= 0) {
          return 1.0 / (1.0 + exp(-x));
        }
        else {
          return exp(x) / (1.0 + exp(x));
        }
      }
      case RELU_FUNC: {
        return max(x, 0.0);
      }
      case RELU_BACK_FUNC: {
        if (x > 0) {
          return y;
        }
        else {
          return 0.0;
        }
      }
      case LOG_FUNC: {
        return log(x + 1e-6);
      }
      case LOG_BACK_FUNC: {
        return y / (x + 1e-6);
      }
      case EXP_FUNC: {
        return exp(x);
      }
      case INV_FUNC: {
        return float(1.0 / x);
      }
      case INV_BACK_FUNC: {
        return -(1.0 / (x * x)) * y;
      }
      case IS_CLOSE_FUNC: {
        return (x - y < 1e-2) && (y - x < 1e-2);
      }
      case MAX_FUNC: {
        if (x > y) {
          return x;
        }
        else {
          return y;
        }
      }
      case POW: {
        return pow(x, y);
      }
      case TANH: {
        return tanh(x);
      }
      default: {
        return x + y;
      }
    }
    
}


__device__ int index_to_position(const int* index, const int* strides, int num_dims) {
  /**
   * Converts a multidimensional tensor index into a single-dimensional position in storage
   * based on strides.
   * Args:
   *    index: index tuple of ints
   *    strides: tensor strides
   *    num_dims: number of dimensions in the tensor, e.g. shape/strides of [2, 3, 4] has 3 dimensions
   *
   * Returns:
   *    int - position in storage
  */
    int position = 0;
    for (int i = 0; i < num_dims; ++i) {
        position += index[i] * strides[i];
    }
    return position;
}

__device__ void to_index(int ordinal, const int* shape, int* out_index, int num_dims) {
  /**
   * Convert an ordinal to an index in the shape. Should ensure that enumerating position 0 ... size of
   * a tensor produces every index exactly once. It may not be the inverse of index_to_position.
   * Args:
   *    ordinal: ordinal position to convert
   *    shape: tensor shape
   *    out_index: return index corresponding to position
   *    num_dims: number of dimensions in the tensor
   *
   * Returns:
   *    None (Fills in out_index)
  */
    int cur_ord = ordinal;
    for (int i = num_dims - 1; i >= 0; --i) {
        int sh = shape[i];
        out_index[i] = cur_ord % sh;
        cur_ord /= sh;
    }
}

__device__ void broadcast_index(const int* big_index, const int* big_shape, const int* shape, int* out_index, int num_dims_big, int num_dims) {
  /**
   * Convert a big_index into big_shape to a smaller out_index into shape following broadcasting rules.
   * In this case it may be larger or with more dimensions than the shape given.
   * Additional dimensions may need to be mapped to 0 or removed.
   *
   * Args:
   *    big_index: multidimensional index of bigger tensor
   *    big_shape: tensor shape of bigger tensor
   *    shape: tensor shape of smaller tensor
   *    nums_big_dims: number of dimensions in bigger tensor
   *    out_index: multidimensional index of smaller tensor
   *    nums_big_dims: number of dimensions in bigger tensor
   *    num_dims: number of dimensions in smaller tensor
   *
   * Returns:
   *    None (Fills in out_index)
  */
    for (int i = 0; i < num_dims; ++i) {
        if (shape[i] > 1) {
            out_index[i] = big_index[i + (num_dims_big - num_dims)];
        } else {
            out_index[i] = 0;
        }
    }
}


__global__ void MatrixMultiplyKernel(
    float* out,
    const int* out_shape,
    const int* out_strides,
    float* a_storage,
    const int* a_shape,
    const int* a_strides,
    float* b_storage,
    const int* b_shape,
    const int* b_strides
) {
  /**
   * Multiply two (compact) matrices into an output (also comapct) matrix. Matrix a and b are both in a batch
   * format, with shape [batch_size, m, n], [batch_size, n, p].
   * Requirements:
   * - All data must be first moved to shared memory.
   * - Only read each cell in a and b once.
   * - Only write to global memory once per kernel.
   * There is guarantee that a_shape[0] == b_shape[0], a_shape[2] == b_shape[1],
   * and out_shape[0] == a_shape[0], out_shape[1] == b_shape[1]
   *
   * Args:
   *   out: compact 1D array of size batch_size x m x p to write the output to
   *   out_shape: shape of the output array
   *   out_strides: strides of the output array
   *   a_storage: compact 1D array of size batch_size x m x n
   *   a_shape: shape of the a array
   *   a_strides: strides of the a array
   *   b_storage: comapct 2D array of size batch_size x n x p
   *   b_shape: shape of the b array
   *   b_strides: strides of the b array
   *
   * Returns:
   *   None (Fills in out array)
   */

    __shared__ float a_shared[TILE][TILE];
    __shared__ float b_shared[TILE][TILE];

    // In each block, we will compute a batch of the output matrix
    // All the threads in the block will work together to compute this batch
    int batch = blockIdx.z;

    int bx = blockIdx.x; int by = blockIdx.y;
    int tx = threadIdx.x; int ty = threadIdx.y;

    int a_batch_stride = a_shape[0] > 1 ? a_strides[0] : 0;
    int b_batch_stride = b_shape[0] > 1 ? b_strides[0] : 0;
    int out_batch_stride = out_shape[0] > 1 ? out_strides[0] : 0;

    int m = out_shape[1];
    int n = a_shape[2];
    int p = out_shape[2];
    /// BEGIN ASSIGN1_2

    // Hints:
    // 1. Compute the row and column of the output matrix this block will compute
    // 2. Compute the position in the output array that this thread will write to
    // 3. Iterate over tiles of the two input matrices, read the data into shared memory
    // 4. Synchronize to make sure the data is available to all threads
    // 5. Compute the output tile for this thread block
    // 6. Synchronize to make sure all threads are done computing the output tile for (row, col)
    // 7. Write the output to global memory

    int row = by * TILE + ty;
    int col = bx * TILE + tx;

    float value = 0.0f;

    for (int tileIdx = 0; tileIdx < (n + TILE - 1) / TILE; ++tileIdx) {
        if (row < m && tileIdx * TILE + tx < n) {
            a_shared[ty][tx] = a_storage[batch * a_batch_stride + row * a_strides[1] + (tileIdx * TILE + tx) * a_strides[2]];
        } else {
            a_shared[ty][tx] = 0.0f;
        }
        if (tileIdx * TILE + ty < n && col < p) {
            b_shared[ty][tx] = b_storage[batch * b_batch_stride + (tileIdx * TILE + ty) * b_strides[1] + col * b_strides[2]];
        } else {
            b_shared[ty][tx] = 0.0f;
        }
        __syncthreads(); 

        for (int k = 0; k < TILE; ++k) {
            value += a_shared[ty][k] * b_shared[k][tx];
        }
        __syncthreads(); 
    }

    if (row < m && col < p) {
        out[batch * out_batch_stride + row * out_strides[1] + col * out_strides[2]] = value;
    }
    /// END ASSIGN1_2
}


__global__ void mapKernel(
    float* out, 
    int* out_shape, 
    int* out_strides, 
    int out_size, 
    float* in_storage, 
    int* in_shape, 
    int* in_strides,
    int shape_size,
    int fn_id
) {
  /**
   * Map function. Apply a unary function to each element of the input array and store the result in the output array.
   * Optimization: Parallelize over the elements of the output array.
   *
   * You may find the following functions useful:
   * - index_to_position: converts an index to a position in a compact array
   * - to_index: converts a position to an index in a multidimensional array
   * - broadcast_index: converts an index in a smaller array to an index in a larger array
   *
   * Args:
   *  out: compact 1D array of size out_size to write the output to
   *  out_shape: shape of the output array
   *  out_strides: strides of the output array
   *  out_size: size of the output array
   *  in_storage: compact 1D array of size in_size
   *  in_shape: shape of the input array
   *  in_strides: strides of the input array
   *  shape_size: number of dimensions in the input and output arrays, assume dimensions are the same
   *  fn_id: id of the function to apply to each element of the input array
   *
   * Returns:
   *  None (Fills in out array)
   */

    int out_index[MAX_DIMS];
    int in_index[MAX_DIMS];
    
    /// BEGIN ASSIGN1_2

    // Hints:
    // 1. Compute the position in the output array that this thread will write to
    // 2. Convert the position to the out_index according to out_shape
    // 3. Broadcast the out_index to the in_index according to in_shape (optional in some cases)
    // 4. Calculate the position of element in in_array according to in_index and in_strides
    // 5. Calculate the position of element in out_array according to out_index and out_strides
    // 6. Apply the unary function to the input element and write the output to the out memory
    
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid > out_size) return;

    to_index(tid, out_shape, out_index, shape_size);

    int out_pos = index_to_position(out_index, out_strides, shape_size);

    broadcast_index(out_index, out_shape, in_shape, in_index, shape_size, shape_size);
    int in_pos = index_to_position(in_index, in_strides, shape_size);

    out[out_pos] = fn(fn_id, in_storage[in_pos]);
    /// END ASSIGN1_2
}


__global__ void reduceKernel(
    float* out,
    int* out_shape,
    int* out_strides,
    int out_size,
    float* a_storage,
    int* a_shape,
    int* a_strides,
    int reduce_dim,
    float reduce_value,
    int shape_size,
    int fn_id
) {
  /**
   * Reduce function. Apply a reduce function to elements of the input array a and store the result in the output array.
   * Optimization:
   * Parallelize over the reduction operation. Each kernel performs one reduction.
   * e.g. a = [[1, 2, 3], [4, 5, 6]], kernel0 computes reduce([1, 2, 3]), kernel1 computes reduce([4, 5, 6]).
   *
   * You may find the following functions useful:
   * - index_to_position: converts an index to a position in a compact array
   * - to_index: converts a position to an index in a multidimensional array
   *
   * Args:
   *  out: compact 1D array of size out_size to write the output to
   *  out_shape: shape of the output array
   *  out_strides: strides of the output array
   *  out_size: size of the output array
   *  a_storage: compact 1D array of size in_size
   *  a_shape: shape of the input array
   *  a_strides: strides of the input array
   *  reduce_dim: dimension to reduce on
   *  reduce_value: initial value for the reduction
   *  shape_size: number of dimensions in the input & output array, assert dimensions are the same
   *  fn_id: id of the reduce function, currently only support add, multiply, and max
   *
   *
   * Returns:
   *  None (Fills in out array)
   */

    // __shared__ double cache[BLOCK_DIM]; // Uncomment this line if you want to use shared memory to store partial results
    int out_index[MAX_DIMS];

    /// BEGIN ASSIGN1_2
    /// TODO
    // 1. Define the position of the output element that this thread or this block will write to
    // 2. Convert the out_pos to the out_index according to out_shape
    // 3. Initialize the reduce_value to the output element
    // 4. Iterate over the reduce_dim dimension of the input array to compute the reduced value
    // 5. Write the reduced value to out memory
    
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid > out_size) return;

    to_index(tid, out_shape, out_index, shape_size);

    int* a_index = out_index;

    int out_pos = index_to_position(out_index, out_strides, shape_size);

    out[out_pos] = reduce_value;

    int reduce_size = a_shape[reduce_dim];

    for (int s = 0; s < reduce_size; ++s) {
        a_index[reduce_dim] = s;
        int a_pos = index_to_position(a_index, a_strides, shape_size);
        
        // out_index[reduce_dim] = s;
        // a_pos = index_to_position(out_index, a_strides, shape_size);
        
        out[out_pos] = fn(fn_id, out[out_pos], a_storage[a_pos]);
    } 
    /// END ASSIGN1_2
}

__global__ void zipKernel(
    float* out,
    int* out_shape,
    int* out_strides,
    int out_size,
    int out_shape_size,
    float* a_storage,
    int* a_shape,
    int* a_strides,
    int a_shape_size,
    float* b_storage, 
    int* b_shape, 
    int* b_strides,
    int b_shape_size,
    int fn_id
) {
  /**
   * Zip function. Apply a binary function to elements of the input array a & b and store the result in the output array.
   * Optimization: Parallelize over the elements of the output array.
   *
   * You may find the following functions useful:
   * - index_to_position: converts an index to a position in a compact array
   * - to_index: converts a position to an index in a multidimensional array
   * - broadcast_index: converts an index in a smaller array to an index in a larger array
   *
   * Args:
   *  out: compact 1D array of size out_size to write the output to
   *  out_shape: shape of the output array
   *  out_strides: strides of the output array
   *  out_size: size of the output array
   *  out_shape_size: number of dimensions in the output array
   *  a_storage: compact 1D array of size in_size
   *  a_shape: shape of the input array
   *  a_strides: strides of the input array
   *  a_shape_size: number of dimensions in the input array
   *  b_storage: compact 1D array of size in_size
   *  b_shape: shape of the input array
   *  b_strides: strides of the input array
   *  b_shape_size: number of dimensions in the input array
   *  fn_id: id of the function to apply to each element of the a & b array
   *
   *
   * Returns:
   *  None (Fills in out array)
   */

    int out_index[MAX_DIMS];
    int a_index[MAX_DIMS];
    int b_index[MAX_DIMS];

    /// BEGIN ASSIGN1_2

    // Hints:
    // 1. Compute the position in the output array that this thread will write to
    // 2. Convert the position to the out_index according to out_shape
    // 3. Calculate the position of element in out_array according to out_index and out_strides
    // 4. Broadcast the out_index to the a_index according to a_shape
    // 5. Calculate the position of element in a_array according to a_index and a_strides
    // 6. Broadcast the out_index to the b_index according to b_shape
    // 7.Calculate the position of element in b_array according to b_index and b_strides
    // 8. Apply the binary function to the input elements in a_array & b_array and write the output to the out memory
    
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid > out_size) return;


    to_index(tid, out_shape, out_index, out_shape_size);

    int out_pos = index_to_position(out_index, out_strides, out_shape_size); // FIXME: necessary?

    broadcast_index(out_index, out_shape, a_shape, a_index, out_shape_size, a_shape_size);
    broadcast_index(out_index, out_shape, b_shape, b_index, out_shape_size, b_shape_size);

    int a_pos = index_to_position(a_index, a_strides, a_shape_size);
    int b_pos = index_to_position(b_index, b_strides, b_shape_size);

    out[out_pos] = fn(fn_id, a_storage[a_pos], b_storage[b_pos]);
    /// END ASSIGN1_2
}


extern "C" {

void MatrixMultiply(
    float* out,
    int* out_shape,
    int* out_strides,
    float* a_storage,
    int* a_shape,
    int* a_strides,
    float* b_storage,
    int* b_shape,
    int* b_strides,
    int batch, int m, int p
) {
    int n = a_shape[2];

    // Allocate device memory
    float *d_out, *d_a, *d_b;
    hipMalloc(&d_a, batch * m * n * sizeof(float));
    hipMalloc(&d_b, batch * n * p * sizeof(float));
    hipMalloc(&d_out, batch * m * p * sizeof(float));

    int *d_out_shape, *d_out_strides, *d_a_shape, *d_a_strides, *d_b_shape, *d_b_strides;
    hipMalloc(&d_out_shape, 3 * sizeof(int));
    hipMalloc(&d_out_strides, 3 * sizeof(int));
    hipMalloc(&d_a_shape, 3 * sizeof(int));
    hipMalloc(&d_a_strides, 3 * sizeof(int));
    hipMalloc(&d_b_shape, 3 * sizeof(int));
    hipMalloc(&d_b_strides, 3 * sizeof(int));

    // Copy data to the device
    hipMemcpy(d_a, a_storage, batch * m * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b_storage, batch * n * p * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_out_shape, out_shape, 3 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_out_strides, out_strides, 3 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_a_shape, a_shape, 3 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_a_strides, a_strides, 3 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b_shape, b_shape, 3 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b_strides, b_strides, 3 * sizeof(int), hipMemcpyHostToDevice);

    int threadsPerBlock = 32;
    dim3 blockDims(threadsPerBlock, threadsPerBlock, 1); // Adjust these values based on your specific requirements
    dim3 gridDims((p + threadsPerBlock - 1) / threadsPerBlock, (m + threadsPerBlock - 1) / threadsPerBlock, batch);
    MatrixMultiplyKernel<<<gridDims, blockDims>>>(
        d_out, d_out_shape, d_out_strides, d_a, d_a_shape, d_a_strides, d_b, d_b_shape, d_b_strides
    );

    // Copy back to the host
    hipMemcpy(out, d_out, batch * m * p * sizeof(float), hipMemcpyDeviceToHost);
    
    hipDeviceSynchronize();

    // Check CUDA execution
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
      fprintf(stderr, "Matmul Error: %s\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }

    // Free memory on device
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);
    hipFree(d_out_shape);
    hipFree(d_out_strides);
    hipFree(d_a_shape);
    hipFree(d_a_strides);
    hipFree(d_b_shape);
    hipFree(d_b_strides);
}

void tensorMap(
    float* out, 
    int* out_shape, 
    int* out_strides, 
    int out_size, 
    float* in_storage, 
    int* in_shape, 
    int* in_strides,
    int in_size,
    int shape_size,
    int fn_id
) {
    float *d_out, *d_in;
    // Allocate device memory
    hipMalloc(&d_out, out_size * sizeof(float));
    hipMalloc(&d_in, in_size * sizeof(float));

    int *d_out_shape, *d_out_strides, *d_in_shape, *d_in_strides;
    hipMalloc(&d_out_shape, shape_size * sizeof(int));
    hipMalloc(&d_out_strides, shape_size * sizeof(int));
    hipMalloc(&d_in_shape, shape_size * sizeof(int));
    hipMalloc(&d_in_strides, shape_size * sizeof(int));

    // Copy data from CPU(host) to GPU(device)
    hipMemcpy(d_in, in_storage, in_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_out_shape, out_shape, shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_out_strides, out_strides, shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_in_shape, in_shape, shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_in_strides, in_strides, shape_size * sizeof(int), hipMemcpyHostToDevice);
    
    int threadsPerBlock = 32;
    int blocksPerGrid = (out_size + threadsPerBlock - 1) / threadsPerBlock;
    mapKernel<<<blocksPerGrid, threadsPerBlock>>>(
      d_out, d_out_shape, d_out_strides, out_size, 
      d_in, d_in_shape, d_in_strides, 
      shape_size, fn_id);
    
    // Copy back to the host
    hipMemcpy(out, d_out, out_size * sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    // Check CUDA execution
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
      fprintf(stderr, "Map Error: %s\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }

    // Free memory on device
    hipFree(d_in);
    hipFree(d_out);
    hipFree(d_out_shape);
    hipFree(d_out_strides);
    hipFree(d_in_shape);
    hipFree(d_in_strides);
}


void tensorZip(
    float* out, 
    int* out_shape, 
    int* out_strides, 
    int out_size,
    int out_shape_size,
    float* a_storage, 
    int* a_shape, 
    int* a_strides,
    int a_size,
    int a_shape_size,
    float* b_storage, 
    int* b_shape, 
    int* b_strides,
    int b_size,
    int b_shape_size,
    int fn_id
) {
    // Allocate device memory
    float *d_out, *d_a, *d_b;
    hipMalloc(&d_a, a_size * sizeof(float));
    hipMalloc(&d_b, b_size * sizeof(float));
    hipMalloc(&d_out, out_size * sizeof(float));

    int *d_out_shape, *d_out_strides, *d_a_shape, *d_a_strides, *d_b_shape, *d_b_strides;
    hipMalloc(&d_out_shape, out_shape_size * sizeof(int));
    hipMalloc(&d_out_strides, out_shape_size * sizeof(int));
    hipMalloc(&d_a_shape, a_shape_size * sizeof(int));
    hipMalloc(&d_a_strides, a_shape_size * sizeof(int));
    hipMalloc(&d_b_shape, b_shape_size * sizeof(int));
    hipMalloc(&d_b_strides, b_shape_size * sizeof(int));

    // Copy data to the device
    hipMemcpy(d_a, a_storage, a_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b_storage, b_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_out_shape, out_shape, out_shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_out_strides, out_strides, out_shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_a_shape, a_shape, a_shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_a_strides, a_strides, a_shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b_shape, b_shape, b_shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b_strides, b_strides, b_shape_size * sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel
    int threadsPerBlock = 32;
    int blocksPerGrid = (out_size + threadsPerBlock - 1) / threadsPerBlock;
    zipKernel<<<blocksPerGrid, threadsPerBlock>>>(
      d_out, d_out_shape, d_out_strides, out_size, out_shape_size,
      d_a, d_a_shape, d_a_strides, a_shape_size,
      d_b, d_b_shape, d_b_strides, b_shape_size,
      fn_id);

    // Copy back to the host
    hipMemcpy(out, d_out, out_size * sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();


    // Check CUDA execution
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
      fprintf(stderr, "Zip Error: %s\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }

    // Free memory on device
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);
    hipFree(d_out_shape);
    hipFree(d_out_strides);
    hipFree(d_a_shape);
    hipFree(d_a_strides);
    hipFree(d_b_shape);
    hipFree(d_b_strides);
}



void tensorReduce(
    float* out, 
    int* out_shape, 
    int* out_strides, 
    int out_size, 
    float* a_storage, 
    int* a_shape, 
    int* a_strides, 
    int reduce_dim, 
    float reduce_value,
    int shape_size,
    int fn_id
) {
    // Allocate device memory
    int a_size = out_size * a_shape[reduce_dim];
    float *d_out, *d_a;
    hipMalloc(&d_out, out_size * sizeof(float));
    hipMalloc(&d_a, a_size * sizeof(float));

    int *d_out_shape, *d_out_strides, *d_a_shape, *d_a_strides;
    hipMalloc(&d_out_shape, shape_size * sizeof(int));
    hipMalloc(&d_out_strides, shape_size * sizeof(int));
    hipMalloc(&d_a_shape, shape_size * sizeof(int));
    hipMalloc(&d_a_strides, shape_size * sizeof(int));

    // Copy data to the device
    hipMemcpy(d_a, a_storage, a_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_out_shape, out_shape, shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_out_strides, out_strides, shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_a_shape, a_shape, shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_a_strides, a_strides, shape_size * sizeof(int), hipMemcpyHostToDevice);
    
    // Launch kernel
    int threadsPerBlock = 32;
    int blocksPerGrid = (out_size + threadsPerBlock - 1) / threadsPerBlock;
    reduceKernel<<<blocksPerGrid, threadsPerBlock>>>(
        d_out, d_out_shape, d_out_strides, out_size, 
        d_a, d_a_shape, d_a_strides, 
        reduce_dim, reduce_value, shape_size, fn_id
    );
    
    // Copy back to the host
    hipMemcpy(out, d_out, out_size * sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    // Check CUDA execution
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
      fprintf(stderr, "Reduce Error: %s\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }

    // Free memory on device
    hipFree(d_a);
    hipFree(d_out);
    hipFree(d_out_shape);
    hipFree(d_out_strides);
    hipFree(d_a_shape);
    hipFree(d_a_strides);
}

}
